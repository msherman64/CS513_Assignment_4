
#include <hip/hip_runtime.h>
#include <stdio.h>

int DIM_LIM = 100;
int MAT_COUNT = 2;
int SEED = 10; //seed for rand

class matrix {
public:
    int row; //number of rows, y
    int col; //number of columns, x
    double* data;
    
    matrix(int columns, int rows) :
        col(columns), row(rows),
        data(new double[col * row])
        {}

    __host__ __device__ double& getdata(int x, int y){
       return data[y * col + x]; //vertical position * row length + pos in row
    };
};


__global__ void d_printMat(matrix *mat)
{   
        int dimxn = mat->col;
        int dimyn = mat->row;
        printf("Dim x %d, Dim y %d\n", dimxn, dimyn);
        for(int y = 0; y<dimyn; y++){
            for(int x = 0; x<dimxn; x++){
                printf("%lf ", mat->getdata(x,y));
            }
            printf("\n");
        }
        printf("\n");
}
__host__ void printMat(matrix *mat)
{   
        int dimxn = mat->col;
        int dimyn = mat->row;
        printf("Dim x %d, Dim y %d\n", dimxn, dimyn);
        for(int y = 0; y<dimyn; y++){
            for(int x = 0; x<dimxn; x++){
                printf("%lf ", mat->getdata(x,y));
            }
            printf("\n");
        }
        printf("\n");
}

matrix** initialize(){
    srand(SEED); //init random gen
    int dim[MAT_COUNT + 1]; //stores matrix sizes
    for(int z = 0; z <= MAT_COUNT; z++){
        dim[z] = rand()%DIM_LIM + 1;//random between 1 and limit
    }

    //declare matrix array as pointer
    matrix **mat = (matrix **)malloc(MAT_COUNT * sizeof(matrix*));
    for(int z = 0; z < MAT_COUNT; z++){
        //each matrix shares a dimension with the previous
        int dimx = dim[z];
        int dimy = dim[z+1];

        mat[z] = new matrix(dimx,dimy); //dimx columns, dimy rows
        for(int x = 0; x<dimx; x++){
            for(int y = 0; y<dimy; y++){
                //TODO change to random double
                mat[z]->getdata(x,y) = 5; //initialize each element
            }
        }
    }
    return mat;
}

int main(){

    matrix **mat_arr = initialize();

    matrix *mat = mat_arr[0];

    printMat(mat);

    matrix *d_mat;
    hipMalloc(&d_mat, sizeof(matrix));
    hipMemcpy(d_mat, mat, sizeof(matrix),
            hipMemcpyHostToDevice);
    double *mat_data;
    hipMalloc(&mat_data, sizeof(double) * mat->col * mat->row);
    hipMemcpy(mat_data, mat->data, sizeof(double) * mat->col * mat->row,
            hipMemcpyHostToDevice);
    hipMemcpy(&(d_mat->data),&mat_data, sizeof(double *),
            hipMemcpyHostToDevice);

    d_printMat<<<1,1>>>(d_mat);
    hipDeviceSynchronize();


}

