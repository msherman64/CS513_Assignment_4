
#include <hip/hip_runtime.h>
#include <stdlib.h> //for rand
#include <iostream> // for cout
#include <vector>  //for vector
using std::cout;

int DIM_LIM = 100;
int MAT_COUNT = 10;

int SEED = 10; //seed for rand

//generate random sized matrices, add pointer to each, to vector
//template <typename T> //handle multiple types
class matrix {
public:
    int row; //number of rows, y
    int col; //number of columns, x
//    std::vector<int> data;
    double* data;

    matrix(int columns, int rows) :
        col(columns), row(rows),
        data(new double[col * row])
        {}

    double& getdata(int x, int y){
       return data[y * col + x]; //vertical position * row length + pos in row
    };
};

matrix** initialize(){
    srand(SEED); //init random gen
    int dim[MAT_COUNT + 1]; //stores matrix sizes
    for(int z = 0; z <= MAT_COUNT; z++){
        dim[z] = rand()%DIM_LIM + 1;//random between 1 and limit
    }

    //declare matrix array as pointer
    matrix **mat = (matrix **)malloc(MAT_COUNT * sizeof(matrix*));
    for(int z = 0; z < MAT_COUNT; z++){
        //each matrix shares a dimension with the previous
        int dimx = dim[z];
        int dimy = dim[z+1];

        mat[z] = new matrix(dimx,dimy); //dimx columns, dimy rows
        for(int x = 0; x<dimx; x++){
            for(int y = 0; y<dimy; y++){
                //TODO change to random double
                mat[z]->getdata(x,y) = 5; //initialize each element
            }
        }
    }
    return mat;
}


int main(){

    matrix **mat = initialize(); //get starting matrix

    //debug by printing size and elements of each matrix in mat
    for(int z = 0; z < MAT_COUNT; z++){
        int dimxn = mat[z]->col;
        int dimyn = mat[z]->row;
        std::cout << dimxn <<" ";
        std::cout << dimyn <<" ";
        std::cout << "\n";
        for(int y = 0; y<dimyn; y++){
            for(int x = 0; x<dimxn; x++){
                cout << mat[z]->getdata(x,y) << " ";
            }
            cout << "\n";
        }
        cout << "\n";
    }



};
