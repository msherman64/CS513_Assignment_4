
#include <hip/hip_runtime.h>
#include <stdlib.h> //for rand
#include <iostream> // for cout
#include <vector>  //for vector
using std::cout;

int DIMX_LIM = 10;
int DIMY_LIM = 10;
int MAT_COUNT = 100;

int SEED = 10; //seed for rand

//generate random sized matrices, add pointer to each, to vector
//template <typename T> //handle multiple types
class matrix {
public:
    int row; //number of rows, y
    int col; //number of columns, x
    std::vector<int> data;

    matrix(int columns, int rows) :
        col(columns), row(rows),
        data(col * row)
        {}

    int& getdata(int y, int x){
       return data[y * col + x]; //vertical position * row length + pos in row
    };
};


int main(){

    srand(SEED); //init random gen

    matrix* mat[MAT_COUNT]; //pointer to pointer to int

    for(int z = 0; z < MAT_COUNT; z++){

        int dimx = rand()%DIMX_LIM + 1; //random between 1 and limit
        int dimy = rand()%DIMY_LIM + 1;

        mat[z] = new matrix(dimx,dimy); //dimx columns, dimy rows
        for(int x = 0; x<dimx; x++){
            for(int y = 0; y<dimy; y++){
                mat[z]->getdata(x,y) = 5;
            }
        }
    }
    
    for(int z = 0; z < MAT_COUNT; z++){
        int dimxn = mat[z]->col;
        int dimyn = mat[z]->row;
        std::cout << dimxn <<" ";
        std::cout << dimyn <<" ";
        std::cout << "\n";
        for(int x = 0; x<dimxn; x++){
            for(int y = 0; y<dimyn; y++){
                cout << mat[z]->getdata(x,y) << " ";
            }
            cout << "\n";
        }
        cout << "\n";
    }

};
