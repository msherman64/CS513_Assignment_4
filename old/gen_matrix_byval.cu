
#include <hip/hip_runtime.h>
#include <stdio.h>

int DIM_LIM = 10;
int MAT_COUNT = 20;
int SEED = 15; //seed for rand
    

class matrix {
public:
    int row; //number of rows, y
    int col; //number of columns, x
    double* data;
    
    __host__ __device__ matrix(int columns, int rows) :
        col(columns), row(rows),
        data(new double[columns * rows])
        {}

    __host__ __device__ double& getdata(int x, int y){
       return data[y * col + x]; //vertical position * row length + pos in row
    };
};

__global__ void d_printMat(matrix *mat)
{   
        int dimxn = mat->col;
        int dimyn = mat->row;
        printf("Dim x %d, Dim y %d\n", dimxn, dimyn);
        for(int y = 0; y<dimyn; y++){
            for(int x = 0; x<dimxn; x++){
                printf("%lf ", mat->getdata(x,y));
            }
            printf("\n");
        }
        printf("\n");
}
__host__ void printMat(matrix *mat)
{   
        int dimxn = mat->col;
        int dimyn = mat->row;
        printf("Dim x %d, Dim y %d\n", dimxn, dimyn);
        for(int y = 0; y<dimyn; y++){
            for(int x = 0; x<dimxn; x++){
                printf("%lf ", mat->getdata(x,y));
            }
            printf("\n");
        }
        printf("\n");
}

__global__ void mat_mult(matrix a, matrix b, matrix *ans){
    if(a.row == b.col){
    
        int iter = a.row; //number of mults needed
        printf("a(%d, %d) b(%d,%d)\n",a.col,a.row,b.col,b.row);
        //ans = new matrix(a.col, b.row);
        printf("result %d rows %d cols\n", ans->row, ans->col);

        for(int x = 0; x < ans->col; x++){
            for(int y = 0; y < ans->row; y++){
                ans->getdata(x,y) = 0; //initialize
                for(int z = 0; z < iter; z++){
                    printf("x %d, y %d\n", x, y);
                    double tmp_a = a.getdata(x,y);
                    double tmp_b = b.getdata(y,x);
                    double tmp_mult = tmp_a * tmp_b;
                    ans->getdata(x,y) += tmp_mult;
                }
                //printf("value at %d %d is %f\n", x,y, ans->getdata(x,y));
            }
        }
    }
    else{
        printf("matrix size mismatch");
    }
};

//note, kernel functions cannot use pass by reference. e.g. duh.
__global__ void test_matrix(matrix **tmp){
    matrix *ans = new matrix(1,1);
    ans->getdata(1,1) = 42;
    printf("row %d, col %d, value %lf\n",
           ans->row, ans->col, ans->getdata(1,1));
    *tmp = ans;
}

matrix** initialize(){
    srand(SEED); //init random gen
    int dim[MAT_COUNT + 1]; //stores matrix sizes
    for(int z = 0; z <= MAT_COUNT; z++){
        dim[z] = rand()%DIM_LIM + 1;//random between 1 and limit
    }

    //declare matrix array as pointer
    matrix **mat = (matrix **)malloc(MAT_COUNT * sizeof(matrix*));
    for(int z = 0; z < MAT_COUNT; z++){
        //each matrix shares a dimension with the previous
        int dimx = dim[z];
        int dimy = dim[z+1];

        mat[z] = new matrix(dimx,dimy); //dimx columns, dimy rows
        for(int x = 0; x<dimx; x++){
            for(int y = 0; y<dimy; y++){
                //TODO change to random double
                mat[z]->getdata(x,y) = 5; //initialize each element
            }
        }
    }
    return mat;
}

matrix* copyMatrixDev(matrix *host){
        matrix *d_mat;
        double *tmp_data;
        hipMalloc(&d_mat, sizeof(matrix));
        hipMemcpy(d_mat, host, sizeof(matrix),
                hipMemcpyHostToDevice);
        hipMalloc(&tmp_data, sizeof(double) * host->col * host->row);
        hipMemcpy(tmp_data, host->data, sizeof(double) * host->col * host->row,
                hipMemcpyHostToDevice);
        hipMemcpy(&(d_mat->data),&tmp_data, sizeof(double *),
                hipMemcpyHostToDevice);
        return d_mat;
}

matrix* hostMultMat(matrix *a, matrix *b){
    matrix *result = new matrix(a->col,b->row);
    matrix *d_result = copyMatrixDev(result);
    delete result;

    hipDeviceSynchronize();
    mat_mult<<<1,1>>>(*a,*b,d_result);

    hipDeviceSynchronize();
    return d_result;

}


int main(){

    matrix **mat_arr = initialize();
    matrix *d_mat[MAT_COUNT];
    double *mat_data[MAT_COUNT];


    for(int i = 0; i < MAT_COUNT; i++){

        hipMalloc(&d_mat[i], sizeof(matrix));
        hipMemcpy(d_mat[i], mat_arr[i], sizeof(matrix),
                hipMemcpyHostToDevice);
        hipMalloc(&mat_data[i], sizeof(double) * mat_arr[i]->col * mat_arr[i]->row);
        hipMemcpy(mat_data[i], mat_arr[i]->data, sizeof(double) * mat_arr[i]->col * mat_arr[i]->row,
                hipMemcpyHostToDevice);
        hipMemcpy(&(d_mat[i]->data),&mat_data[i], sizeof(double *),
                hipMemcpyHostToDevice);
    //    printMat(mat_arr[i]);
    //    d_printMat<<<1,1>>>(d_mat[i]);
    }

        //matrix *d_result = hostMultMat(mat_arr[0], d_mat[0], mat_arr[1], d_mat[1]);
        //d_printMat<<<1,1>>>(d_result);
        //cudaDeviceSynchronize();


    matrix *d_result;
//    for(int i = 0; i < MAT_COUNT-1; i++){
    for(int i = 0; i < 1; i++){
        d_result = hostMultMat(mat_arr[i], mat_arr[i+1]);
        d_printMat<<<1,1>>>(d_result);
        hipFree(d_result);

    }




}

